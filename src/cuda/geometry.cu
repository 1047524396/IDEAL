#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../geometry/MyPolygon.h"
#include "mygpu.h"
#include "cuda_util.h"

// return the distance of two segments

__device__
bool check_contain(const double *polygon1, const double *polygon2, int num_vertices_1, int num_vertices_2){
	bool val = false;
	for(int p = 0;p<num_vertices_2-1;p++){
		double px = polygon2[p];
		double py = polygon2[num_vertices_2+p];
		for (int i = 0, j = 1; i < num_vertices_1-1; i++,j++) {
			// segment i->j intersect with line y=p.y
			double pyi = polygon1[num_vertices_1+i];
			double pyj = polygon1[num_vertices_1+j];
			if ((pyj>py) != (pyi>py))
			{
				double pxi = polygon1[i];
				double pxj = polygon1[j];
				double a = (pxj-pxi) / (pyj-pyi);
				if(px-pxi<a*(py-pyi)){
					val = !val;
				}
			}
		}
	}
	return val;
}

__global__
void contain_cuda(const double *poly1, const double *poly2, const uint *offset_size, int *ret, size_t num_pairs){

	// which polygon-polygon pair
	int pair_id = blockIdx.x*blockDim.x+threadIdx.x;
	if(pair_id>=num_pairs){
		return;
	}

	uint off1 = offset_size[pair_id*4];
	uint size1 = offset_size[pair_id*4+1];
	uint off2 = offset_size[pair_id*4+2];
	uint size2 = offset_size[pair_id*4+3];
//	printf("os: %d %d %d %d\n",off1, size1, off2,size2);
//
//	for(int i=0;i<size1;i++){
//		//printf("%f %f\n",(poly1+off1)[i],(poly1+off1)[i+size1]);
//	}
//	for(int i=0;i<size2;i++){
//		printf("%f %f\n",(poly2+off2)[i],(poly2+off2)[i+size1]);
//	}
	//printf("%d %d %d %d\n",off1, size1, off2, size2);
	ret[pair_id] = (int)check_contain(poly1+off1,poly2+off2,size1,size2);
}

/*
 * data: contains the segments of the meshes mentioned in this join.
 * offset_size:  contains the offset in the data for each batch, and the sizes of two data sets
 * result: for the returned results for each batch
 * batch_num: number of computed batches
 *
 * */
void contain_batch_gpu(gpu_info *gpu, double *data, uint *offset_size, int *result, size_t total_vertice_num, int pair_num){

	assert(gpu);
	hipSetDevice(gpu->device_id);
	struct timeval start = get_cur_time();

	// space for the results in GPU
	int *d_ret = gpu->get_result(sizeof(int)*pair_num);
	// space for the offset and size information in GPU
	uint *d_os = gpu->get_os(sizeof(uint)*pair_num*4);
	double *d_poly1 = gpu->source_data;
	double *d_poly2 = gpu->get_data(total_vertice_num*2*sizeof(double));

	CUDA_SAFE_CALL(hipMemcpy(d_poly2, data, total_vertice_num*2*sizeof(double), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_os, offset_size, pair_num*4*sizeof(uint), hipMemcpyHostToDevice));
	//logt("allocating data", start);

	// compute the vectors of segments in data, save to d_vec
	contain_cuda<<<pair_num/1024+1,1024>>>(d_poly1, d_poly2, d_os, d_ret, pair_num);
	check_execution();
	hipDeviceSynchronize();
	//logt("computations", start);
	CUDA_SAFE_CALL(hipMemcpy(result, d_ret, pair_num*sizeof(int), hipMemcpyDeviceToHost));
	//logt("copy data out", start);
}

void load_source_togpu(gpu_info *gpu, vector<MyPolygon *> &source){
	size_t source_size = 0;
	for(MyPolygon *p:source){
		source_size += 2*p->boundary->num_vertices*sizeof(double);
	}
	gpu->get_source(source_size);
	source_size = 0;
	for(MyPolygon *p:source){
		int num_vertices = p->boundary->num_vertices;
		p->offset = source_size/sizeof(double);
		CUDA_SAFE_CALL(hipMemcpy((char *)(gpu->source_data)+source_size, (char *)(p->boundary->x), num_vertices*sizeof(double),hipMemcpyHostToDevice));
		source_size += num_vertices*sizeof(double);
		CUDA_SAFE_CALL(hipMemcpy((char *)(gpu->source_data)+source_size, (char *)(p->boundary->y), num_vertices*sizeof(double),hipMemcpyHostToDevice));
		source_size += num_vertices*sizeof(double);
	}
}


